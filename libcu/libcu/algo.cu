#include "hip/hip_runtime.h"

#include <hip/device_functions.h>

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#include "libcu.h"

__global__ void cu_multiply(real input[], real weight[], real merge[], size_t input_num, size_t weight_num)
{
	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	while (thread_idx < weight_num)
	{
		merge[thread_idx] = input[thread_idx % input_num] * weight[thread_idx];

		thread_idx += thread_num;
	}
}

__global__ void cu_merge_output(real merge[], size_t input_num, size_t merge_num, size_t remainder, size_t output_num)
{
	size_t oprand_num = merge_num * output_num;
	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	while (thread_idx < oprand_num)
	{
		size_t neuro_idx = thread_idx / merge_num;
		size_t weight_idx = thread_idx % merge_num;
		size_t base_idx = neuro_idx * input_num + weight_idx;

		merge[base_idx] += merge[base_idx + merge_num + remainder];

		thread_idx += thread_num;
	}
}

__global__ void cu_output(real merge[], real bias[], real output[], size_t input_num, size_t output_num)
{
	//const static real TWO_DIV_THREE = (real)(2.0 / 3.0);

	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	while (thread_idx < output_num)
	{
		real out = merge[thread_idx * input_num] + bias[thread_idx];
		output[thread_idx] = ((real)2.0 * tanh(out * 0.01));
		//output[thread_idx] = ((real)1.7159 * tanh(out * TWO_DIV_THREE));
		//real tout = ((real)1.7159 * tanh(out * TWO_DIV_THREE));
		//if (__isnan(tout))
		//{
		//	tout = out > 0 ? (real)1.7159 : (real)-1.7159;
		//}
		//output[thread_idx] = tout;

		thread_idx += thread_num;
	}
}

#define THREAD_NUM_PER_BLOCK (1024)
size_t get_block_num(size_t oprand_num)
{
	const static size_t max_block_num = 42 * 1024;
	oprand_num = oprand_num / THREAD_NUM_PER_BLOCK;
	++oprand_num;
	return oprand_num < max_block_num ? oprand_num : max_block_num;
}

void calculate_merge(real merge[], size_t block_size, size_t block_num)
{
	size_t remainder = block_size & 1;
	size_t merge_num = block_size >> 1;

	while (merge_num)
	{
		cu_merge_output<<<get_block_num(merge_num * block_num), THREAD_NUM_PER_BLOCK >>>(merge, block_size, merge_num, remainder, block_num);

		remainder = (merge_num += remainder) & 1;
		
		merge_num >>= 1; // merge_num /= 2;
	}
}

void calculate_layer_output(real input[], real weight[], real bias[], real output[], size_t input_num, size_t output_num, real merge[])
{
	cu_multiply<<<get_block_num(input_num * output_num), THREAD_NUM_PER_BLOCK >>>
		(input, weight, merge, input_num, input_num * output_num);
	calculate_merge(merge, input_num, output_num);
	cu_output<<<get_block_num(output_num), THREAD_NUM_PER_BLOCK >>>
		(merge, bias, output, input_num, output_num);
}

__global__ void cu_train_weight(real input[], real grad[], real weight[], size_t input_num, size_t weight_num, real study_rate)
{
	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	while (thread_idx < weight_num)
	{
		size_t neuro_idx = thread_idx / input_num;
		size_t weight_idx = thread_idx % input_num;

		weight[thread_idx] += input[weight_idx] * grad[neuro_idx];// *study_rate;

		thread_idx += thread_num;
	}
}

__global__ void cu_train_bias(real grad[], real bias[], size_t output_num, real study_rate)
{
	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	while (thread_idx < output_num)
	{
		bias[thread_idx] += grad[thread_idx];// *study_rate;

		thread_idx += thread_num;
	}
}

__global__ void cu_target_to_grad(real output[], real target[], real grad[], real study_rate, size_t output_num)
{
	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	while (thread_idx < output_num)
	{
		grad[thread_idx] = (target[thread_idx] - output[thread_idx]) * study_rate;

		thread_idx += thread_num;
	}
}

__global__ void cu_grad_to_grad_merge(real gradin[], real weight[], real merge[], size_t input_num, size_t output_num)
{
	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	size_t weight_num = input_num * output_num;
	while (thread_idx < weight_num)
	{
		size_t neuro_idx = thread_idx / input_num;
		size_t weight_idx = thread_idx % input_num;
		merge[weight_idx * output_num + neuro_idx] = gradin[neuro_idx] * weight[thread_idx];

		thread_idx += thread_num;
	}
}

__global__ void cu_merge_to_grad(real merge[], real grad[], size_t input_num, size_t output_num)
{
	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	while (thread_idx < input_num)
	{
		grad[thread_idx] = merge[thread_idx * output_num];

		thread_idx += thread_num;
	}
}

void calculate_layer_grad(real output[], real target[], real grad[], real study_rate, size_t output_num)
{
	cu_target_to_grad<<<get_block_num(output_num), THREAD_NUM_PER_BLOCK >>>
		(output, target, grad, study_rate, output_num);
}

void calculate_layer_grad(real gradin[], real weight[], real grad[], size_t input_num, size_t output_num, real merge[])
{
	cu_grad_to_grad_merge<<<get_block_num(input_num * output_num), THREAD_NUM_PER_BLOCK >>>
		(gradin, weight, merge, input_num, output_num);
	calculate_merge(merge, output_num, input_num);
	cu_merge_to_grad<<<get_block_num(input_num), THREAD_NUM_PER_BLOCK >>>
		(merge, grad, input_num, output_num);
}

void calculate_layer_train(real input[], real grad[], real weight[], real bias[], size_t input_num, size_t output_num, real study_rate)
{
	cu_train_weight<<<get_block_num(input_num * output_num), THREAD_NUM_PER_BLOCK >>>
		(input, grad, weight, input_num, input_num * output_num, study_rate);
	cu_train_bias<<<get_block_num(output_num), THREAD_NUM_PER_BLOCK >>>
		(grad, bias, output_num, study_rate);
}

template<typename srctype, typename dsttype>
__global__ void cuda_translate_data_format(void *dst, const void *src, size_t pixelNum)
{
	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	while (thread_idx < pixelNum)
	{
		if ((thread_idx & 0x03) != 3)
		{
			((dsttype *)dst)[thread_idx] = (((dsttype)((const srctype *)src)[((thread_idx >> 2) * 3) + (thread_idx & 0x03)]) + (dsttype)1.0) * (dsttype)0.5;
		}
		else
		{
			((dsttype *)dst)[thread_idx] = (dsttype)0;
		}
		thread_idx += thread_num;
	}
}

void translate_data_format(void *dst, const void *src, size_t pixelNum, enum_translate_type type)
{
	pixelNum *= 4;
	switch (type)
	{
	case float_to_double:
		cuda_translate_data_format<float, double><<<get_block_num(pixelNum), THREAD_NUM_PER_BLOCK >>>(dst, src, pixelNum);
		break;
	case double_to_float:
		cuda_translate_data_format<double, float><<<get_block_num(pixelNum), THREAD_NUM_PER_BLOCK >>>(dst, src, pixelNum);
		break;
	case real_to_double:
		cuda_translate_data_format<real, double><<<get_block_num(pixelNum), THREAD_NUM_PER_BLOCK >>>(dst, src, pixelNum);
		break;
	case real_to_float:
		cuda_translate_data_format<real, float><<<get_block_num(pixelNum), THREAD_NUM_PER_BLOCK >>>(dst, src, pixelNum);
		break;
	case double_to_real:
		cuda_translate_data_format<double, real><<<get_block_num(pixelNum), THREAD_NUM_PER_BLOCK >>>(dst, src, pixelNum);
		break;
	case float_to_real:
		cuda_translate_data_format<float, real><<<get_block_num(pixelNum), THREAD_NUM_PER_BLOCK >>>(dst, src, pixelNum);
		break;
	default:
		break;
	}
}

__global__ void cuda_set_value(real data[], size_t count, real val)
{
	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	while (thread_idx < count)
	{
		data[thread_idx] = val;
		thread_idx += thread_num;
	}
}

void set_value(real data[], size_t count, real val)
{
	cuda_set_value<<<get_block_num(count), THREAD_NUM_PER_BLOCK >>> (data, count, val);
}

__device__ size_t cuda_get_rand_value(size_t seed)
{
	const static size_t a = 701507, b = 696497;
	char *p = (char *)&seed;
	size_t hash = a;
	hash ^= p[0];
	hash *= b;
	hash ^= p[1];
	hash *= b;
	hash ^= p[2];
	hash *= b;
	hash ^= p[3];
	hash *= b;
	return hash;
}

__global__ void cuda_set_rand_value(real data[], size_t count)
{
	size_t thread_num = gridDim.x * blockDim.x;
	size_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;
	while (thread_idx < count)
	{
		data[thread_idx] = (real)((cuda_get_rand_value((size_t)&data[count]) & 0x0FFFFFF) - 0x07FFFFF) / (real)(0x07FFFFF);
		thread_idx += thread_num;
	}
}

void set_rand_value(real data[], size_t count)
{
	cuda_set_rand_value<<<get_block_num(count), THREAD_NUM_PER_BLOCK >>> (data, count);
}

/*#include <iostream>

int main()
{
	const static size_t in_num = 96 * 96 * 4;
	const static size_t out_num = 1024 * 2;
	const static size_t wei_num = in_num * out_num;

	cu_array<real> in(in_num), wei(wei_num), bias(out_num), tar(out_num);
	cu_array<real> out(out_num), grd(out_num), mrg(wei_num);
	
	for (size_t i = 0; i < in_num; ++i)
	{
		in[i] = ((real) rand() / (real) RAND_MAX);
	}
	in.set();
	for (size_t i = 0; i < wei_num; ++i)
	{
		wei[i] = ((real) rand() / (real) RAND_MAX);
	}
	wei.set();
	for (size_t i = 0; i < out_num; ++i)
	{
		bias[i] = ((real) rand() / (real) RAND_MAX);
		tar[i] = (real) i / (real) out_num;
	}
	bias.set();
	tar.set();

	calculate_layer_output(in(), wei(), bias(), out(), in_num, out_num, mrg());
	for (int i = 0; i < 1000000; ++i)
	{
		calculate_layer_grad(out(), tar(), grd(), out_num);
		calculate_layer_train(in(), grd(), wei(), bias(), in_num, out_num, 0.0001);
		calculate_layer_output(in(), wei(), bias(), out(), in_num, out_num, mrg());

		if ((i + 1) % 10 == 0)
		{
			out.get();
			for (int j = 0; j < out_num; ++j)
			{
				printf(" %15f", tar[j] - out[j]);
			}
			printf("\n\n");
		}
	}

	system("pause");
	return 0;
}*/
