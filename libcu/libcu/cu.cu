
#include "libcu.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

void set_cuda_device(int dev)
{
	hipSetDevice(dev);
}

void *cuda_malloc(size_t size)
{
	void *ptr = 0;
	hipMalloc(&ptr, size);
	return ptr;
}

void *cuda_malloc_host(size_t size)
{
	void *ptr = 0;
	hipHostMalloc(&ptr, size);
	return ptr;
}

void cuda_free(void *ptr)
{
	hipFree(ptr);
}

void cuda_memcpy(void *dst, const void *src, size_t size, enum_cuda_memcpy_direction direction)
{
	hipMemcpy(dst, src, size, (hipMemcpyKind)direction);
}

/*void cuda_host_to_host(void *dst, const void *src, size_t size)
{
	hipMemcpy(dst, src, size, hipMemcpyHostToHost);
}

void cuda_host_to_device(void *dst, const void *src, size_t size)
{
	hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void cuda_device_to_host(void *dst, const void *src, size_t size)
{
	hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void cuda_device_to_device(void *dst, const void *src, size_t size)
{
	hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
}*/

const char *cuda_get_last_error()
{
	return hipGetErrorString(hipGetLastError());
}
