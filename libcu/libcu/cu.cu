
#include "libcu.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

void *cuda_malloc(size_t size)
{
	void *ptr = 0;
	hipMalloc(&ptr, size);
	return ptr;
}

void *cuda_malloc_host(size_t size)
{
	void *ptr = 0;
	hipHostMalloc(&ptr, size);
	return ptr;
}

void cuda_free(void *ptr)
{
	hipFree(ptr);
}

void cuda_host_to_host(void *dst, void *src, size_t size)
{
	hipMemcpy(dst, src, size, hipMemcpyHostToHost);
}

void cuda_host_to_device(void *dst, void *src, size_t size)
{
	hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void cuda_device_to_host(void *dst, void *src, size_t size)
{
	hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void cuda_device_to_device(void *dst, void *src, size_t size)
{
	hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
}
