
#include "libcu.h"

#pragma comment(lib, "opengl32.lib")
#pragma comment(lib, "glu32.lib")

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <stdio.h>

bool gl_set_texture(GLuint texIdx, void *data, GLuint size, char *errmsg)
{
	hipError_t err;

	hipGraphicsResource_t resource;
	if ((err = hipGraphicsGLRegisterImage(&resource, texIdx, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard)) != hipSuccess)
	{
		if (errmsg)
		{
			sprintf(errmsg, "<%s>%s", "hipGraphicsGLRegisterImage", hipGetErrorString(err));
		}
		return false;
	}

	if ((err = hipGraphicsMapResources(1, &resource)) != hipSuccess)
	{
		if (errmsg)
		{
			sprintf(errmsg, "<%s>%s", "hipGraphicsMapResources", hipGetErrorString(err));
		}
		hipGraphicsUnregisterResource(resource);
		return false;
	}

	hipArray_t dst;
	if ((err = hipGraphicsSubResourceGetMappedArray(&dst, resource, 0, 0)) != hipSuccess)
	{
		if (errmsg)
		{
			sprintf(errmsg, "<%s>%s", "hipGraphicsSubResourceGetMappedArray", hipGetErrorString(err));
		}
		hipGraphicsUnmapResources(1, &resource);
		hipGraphicsUnregisterResource(resource);
		return false;
	}

	if ((err = hipMemcpyToArray(dst, 0, 0, data, size, hipMemcpyDeviceToDevice)) != hipSuccess)
	{
		if (errmsg)
		{
			sprintf(errmsg, "<%s>%s", "hipMemcpyToArray", hipGetErrorString(err));
		}
		hipGraphicsUnmapResources(1, &resource);
		hipGraphicsUnregisterResource(resource);
		return false;
	}

	hipGraphicsUnmapResources(1, &resource);
	hipGraphicsUnregisterResource(resource);
	return true;
}
