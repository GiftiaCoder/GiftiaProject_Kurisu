
#include "libcu.h"

#pragma comment(lib, "opengl32.lib")
#pragma comment(lib, "glu32.lib")

#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <stdio.h>

static hipError_t gl_set_texture(GLuint texIdx, const void *data, size_t size, hipGraphicsResource_t *resource, char *errmsg)
{
	hipError_t err;
	
	if ((err = hipGraphicsGLRegisterImage(resource, texIdx, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard)) != hipSuccess)
	{
		if (errmsg)
		{
			sprintf(errmsg, "<%s>%s", "hipGraphicsGLRegisterImage", hipGetErrorString(err));
		}
		return err;
	}

	if ((err = hipGraphicsMapResources(1, resource)) != hipSuccess)
	{
		if (errmsg)
		{
			sprintf(errmsg, "<%s>%s", "hipGraphicsMapResources", hipGetErrorString(err));
		}
		return err;
	}

	hipArray_t dst = 0;
	if ((err = hipGraphicsSubResourceGetMappedArray(&dst, *resource, 0, 0)) != hipSuccess)
	{
		if (errmsg)
		{
			sprintf(errmsg, "<%s>%s", "hipGraphicsSubResourceGetMappedArray", hipGetErrorString(err));
		}
		return err;
	}

	//hipChannelFormatDesc d;
	//hipExtent e;
	//unsigned int f;
	//hipArrayGetInfo(&d, &e, &f, dst);
	//printf("(%d, %d, %d)(%d, %d, %d, %d), %d\n", e.width, e.height, e.depth, d.x, d.y, d.z, d.w, d.f);

	if ((err = hipMemcpyToArray(dst, 0, 0, data, size, hipMemcpyDeviceToDevice)) != hipSuccess)
	{
		if (errmsg)
		{
			sprintf(errmsg, "<%s>%s", "hipMemcpyToArray", hipGetErrorString(err));
		}
		return err;
	}

	return err;
}

bool gl_set_texture(GLuint texIdx, const void *data, size_t size, char *errmsg)
{
	hipDeviceSynchronize();

	hipGraphicsResource_t resource = 0;
	
	bool ret = (gl_set_texture(texIdx, data, size, &resource, errmsg) == hipSuccess);

	hipGraphicsUnmapResources(1, &resource);
	if (resource)
	{
		hipGraphicsUnregisterResource(resource);
	}

	return ret;
}
